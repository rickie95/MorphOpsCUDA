#include "hip/hip_runtime.h"
#include "MorphableOperator.h"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
		hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err){
	if (err == hipSuccess)
			return;
		std::cerr << statement <<
				" returned " <<
				hipGetErrorString(err) << "("
				<< err << ") at " << file << ":" << line << std::endl;
		exit(1);
}

__host__ Image_t* opening(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){ // EROSION then DILATATION
	std::chrono::duration<double> erosion_time, dilatation_time;
    Image_t* opened = dilatation(erosion(input, structElem, &erosion_time), structElem, &dilatation_time);
    if(time_span != NULL)
        *time_span = std::chrono::duration_cast<std::chrono::duration<double>>(erosion_time + dilatation_time);
    return opened;
}

__host__ Image_t* closing(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){
	std::chrono::duration<double> erosion_time, dilatation_time;
    Image_t* closed = erosion(dilatation(input, structElem, &dilatation_time), structElem, &erosion_time);
    if(time_span != NULL)
    	*time_span = std::chrono::duration_cast<std::chrono::duration<double>>(erosion_time + dilatation_time);
    return closed;
}

__host__ Image_t* topHat(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){
	// Originale - Apertura
	Image_t* opened = opening(input, structElem, time_span);
	float *topHat_data = (float*)malloc(input->width * input->height * sizeof(float));
	for(int i = 0; i < input->width * input->height; i += 1) // Maybe should transposed on GPU?
		topHat_data[i] = max_pixel(opened->data[i] - input->data[i], 0);

	Image_delete(opened);
	return Image_new(input->width, input->height, 1, topHat_data);
}

__host__ Image_t* bottomHat(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){
	// Chiusura - originale
	Image_t* closed = closing(input, structElem, time_span);
	float *bottomHat_data = (float*)malloc(input->width * input->height * sizeof(float));
	for(int i = 0; i < input->width * input->height; i += 1) // Maybe should transposed on GPU?
	    bottomHat_data[i] = max_pixel(input->data[i] + closed->data[i], 0);

	Image_delete(closed);
	return Image_new(input->width, input->height, 1, bottomHat_data);
}

__host__ Image_t* erosion(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){
	// malloc for I/O images and SE
	float *deviceInputImage, *deviceOutputImage, *deviceSEData, *hostOutputImage=NULL;

	std::chrono::high_resolution_clock::time_point t_start, t_end;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&deviceInputImage, sizeof(float) * input->height * input->width));
	CUDA_CHECK_RETURN(hipMalloc((void ** )&deviceOutputImage, sizeof(float) * input->height * input->width));

	// Send data (Input and SE)
	CUDA_CHECK_RETURN(hipMemcpy(deviceInputImage, input->data, input->height * input->width * sizeof(float),hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(deviceSEdata), structElem->data, structElem->get_width() * structElem->get_height() * sizeof(float)));
	// COMPUTE
	t_start = std::chrono::high_resolution_clock::now();
	dim3 dimGrid(ceil((float) input->width / TILE_WIDTH), ceil((float) input->height / TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int shared_amount = (TILE_WIDTH + structElem->get_width() - 1) * (TILE_WIDTH + structElem->get_height() - 1) * sizeof(float);
	process<<<dimGrid, dimBlock, shared_amount>>>(deviceInputImage, deviceOutputImage, input->width, input->height, deviceSEData,
		structElem->get_width(), structElem->get_height(), EROSION);
	hipDeviceSynchronize(); // wait for completion
	t_end = std::chrono::high_resolution_clock::now();
	if(time_span != NULL)
	    *time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t_end - t_start);

    hostOutputImage = (float*)malloc(input->height * input->width * sizeof(float));
	// download data
	CUDA_CHECK_RETURN(hipMemcpy(hostOutputImage, deviceOutputImage, input->height * input->width * sizeof(float),hipMemcpyDeviceToHost));
    Image_t *output = Image_new(input->width, input->height, 1, hostOutputImage);
	// free memory on GPU
	hipFree(deviceInputImage);
	hipFree(deviceOutputImage);
	hipFree(deviceSEData);
    return output;
}

__host__ Image_t* dilatation(Image_t* input, StructElem* structElem, std::chrono::duration<double> *time_span){
// malloc for I/O images and SE
    float *deviceInputImage, *deviceOutputImage, *deviceSEData, *hostOutputImage=NULL;

    std::chrono::high_resolution_clock::time_point t_start, t_end;
    CUDA_CHECK_RETURN(hipMalloc((void ** )&deviceInputImage, sizeof(float) * input->height * input->width));
    CUDA_CHECK_RETURN(hipMalloc((void ** )&deviceOutputImage, sizeof(float) * input->height * input->width));
    // Send data (Input and SE)
    CUDA_CHECK_RETURN(hipMemcpy(deviceInputImage, input->data, input->height * input->width * sizeof(float),hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(deviceSEdata), structElem->data, structElem->get_width() * structElem->get_height() * sizeof(float)));

    // COMPUTE
    t_start = std::chrono::high_resolution_clock::now();
    dim3 dimGrid(ceil((float) input->width / TILE_WIDTH), ceil((float) input->height / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    int shared_amount = (TILE_WIDTH + structElem->get_width() - 1) * (TILE_WIDTH + structElem->get_height() - 1) * sizeof(float);
    process<<<dimGrid, dimBlock, shared_amount>>>(deviceInputImage, deviceOutputImage, input->width, input->height, deviceSEData,
    structElem->get_width(), structElem->get_height(), DILATATION);
    hipDeviceSynchronize(); // wait for completion
    t_end = std::chrono::high_resolution_clock::now();
    *time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t_end - t_start);
    hostOutputImage = (float*)malloc(input->height * input->width * sizeof(float));
    // download data
    CUDA_CHECK_RETURN(hipMemcpy(hostOutputImage, deviceOutputImage, input->height * input->width * sizeof(float),hipMemcpyDeviceToHost));
    Image_t *output = Image_new(input->width, input->height, 1, hostOutputImage);
    // free memory
    hipFree(deviceInputImage);
    hipFree(deviceOutputImage);
    hipFree(deviceSEData);
    return output;
}

__global__ void process(float *input_img, float *output_img, int img_W, int img_H,
		const float *__restrict__ SE,const int SE_W, const int SE_H, int operation) {

	// Assumes that input_img->data is a NxM float matrix with values 0 or 1
	// OPERATION: 1 if erosion, 0 if dilatation

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int w = TILE_WIDTH + SE_W - 1;
	extern __shared__ float input_ds[];
	int dest = threadIdx.y * w + threadIdx.x;
	int destY = dest / w;
	int destX = dest % w;
	int globalX = blockIdx.x * TILE_WIDTH + tx;
	int globalY = blockIdx.y * TILE_WIDTH + ty;
	int globalCoord = globalY * img_W + globalX;

	// LOAD INNER TILE
	if (globalY >= 0 && globalY < img_H && globalX >= 0 && globalX < img_W) {
		input_ds[(destY + SE_H / 2) * w + destX + SE_W / 2] = input_img[globalCoord];
	} else {
		input_ds[(destY + SE_H / 2) * w + destX + SE_W / 2] = -1;
	}

	__syncthreads();

	if (ty == 0)  // TOP COLUMN
		for (int row = 0; row < SE_H / 2; row += 1)
			if (globalY == 0) {
				input_ds[row * w + destX + SE_W / 2] = -1
			} else {
				input_ds[row * w + destX + SE_W / 2] = input_img[(globalY - SE_H / 2 + row) * img_W + globalX];
			}

	if (tx == 0)  // LEFT COLUMN
		for (int col = 0; col < SE_W / 2; col += 1)
			if (globalX == 0) {
				input_ds[(destY + SE_H / 2) * w + col] = -1
			} else {
				input_ds[(destY + SE_H / 2) * w + col] = input_img[globalY * img_W + (globalX - SE_W / 2 + col)];
			}

	if(ty == TILE_WIDTH - 1) // BOTTOM COLUMN
		for (int row = w - SE_H / 2; row < w; row += 1) {
			int gy = globalY + (row - w + SE_H / 2);
			if (gy >= img_H) {
				input_ds[row * w + destX + SE_W / 2] = -1;
			} else {
				input_ds[row * w + destX + SE_W / 2] = input_img[gy * img_W + globalX];
			}
		}

	if(tx == TILE_WIDTH - 1) // RIGHT ROW
		for(int col = SE_W/2; col < w; col += 1){
			int gx = globalX - w + SE_H/2 + col;
			if(gx >= img_W){
				input_ds[(destY + SE_H/2) * w + col] = -1;
			} else{
				input_ds[(destY + SE_H/2) * w + col] = input_img[globalY * img_W + gx];
			}
		}

	if(tx == 0 && ty == 0) // NW CORNER
		for(int row = 0; row < SE_H/2; row += 1)
			for(int col = 0; col < SE_W/2; col += 1) {
                int gx = globalX - SE_W/2 + col;
                int gy = globalY - SE_H/2 + row;
                if (gx < 0 || gy < 0) { // Every pixel on the left border or the top border
                    input_ds[row * w + col] = -1;
                } else {
                    input_ds[row * w + col] = input_img[gy * img_W + gx];
                }
            }

	if(tx == TILE_WIDTH - 1 && ty == 0) // NE CORNER
        for(int row = 0; row < SE_H/2; row += 1)
            for(int col = TILE_WIDTH + SE_H/2; col < w; col += 1) {
                int gx = globalX + 1 + col;
                int gy = globalY - SE_H / 2 + row;
                if (gx >= img_W || gy < 0) { // Every tile on the top or the right border
                    input_ds[row * w + col] = -1;
                } else {
                    input_ds[row * w + col] = input_img[gy * img_W + gx];
                }
            }

	if(tx == 0 && ty == TILE_WIDTH - 1) // SW CORNER
	    for(int row = TILE_WIDTH + SE_H/2; row < w; row += 1)
	        for(int col = 0; col < SE_W/2; col += 1){
                int gx = globalX - SE_W/2 + col;
                int gy = globalY + 1  + row;
                if(gx < 0 || gy >= img_H ){
                    input_ds[row * w + col] = -1;
                } else {
                    input_ds[row * w + col] = input_img[gy * img_W + gx];
                }
            }

	if(tx == TILE_WIDTH - 1 && ty == TILE_WIDTH - 1) // SE CORNER
	     for(int row = TILE_WIDTH + SE_H/2; row < w; row += 1)
	         for(int col = TILE_WIDTH + SE_W/2; col < w; col += 1){
                 int gx = globalX + 1 + col;
                 int gy = globalY + 1 + row;
                 if(gx >= img_W || gy >= img_H){
                     input_ds[row * w + col] = -1;
                 } else {
                     input_ds[row * w + col] = input_img[gy * img_W + gx];
                 }
	      }
	
    __syncthreads();
    // 2) COMPUTE - load neighborhood and write max/min
    if((globalY >= 0 && globalY < img_H && globalX >= 0 && globalX < img_W)){
        float max = input_ds[(destY + SE_H/2)*w + destX + SE_W/2];
        float min = max;

    	for(int row = - SE_H/2 ; row < SE_H/2; row+=1){
    		for(int col = -SE_W/2; col < SE_W/2; col+=1){
    			if(deviceSEdata[(row + SE_H/2) * SE_W + col + SE_W/2] > 0 &&
    					input_ds[(destY + SE_H/2 + row) * w + destX + SE_W/2 + col] > -1){
    				if (max < input_ds[(destY + SE_H/2 + row) * w + destX + SE_W/2 + col])
    					max = input_ds[(destY + SE_H/2 + row) * w + destX + SE_W/2 + col];
    				if (min > input_ds[(destY + SE_H/2 + row) * w + destX + SE_W/2 + col])
    					min = input_ds[(destY + SE_H/2 + row) * w + destX + SE_W/2 + col];
    			}
    		}
    	}
    	// Write value

    	if(operation == EROSION)
    	    output_img[globalCoord] = max;
    	if(operation == DILATATION)
    		output_img[globalCoord] = min;

    }
    __syncthreads();

}

__host__ float max_pixel(float a, float b){
	return a>b ? a : b;
}

__host__ float min_pixel(float a, float b){
	return a<b ? a : b;
}


